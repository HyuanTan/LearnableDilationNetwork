#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/layers/stop_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void StopForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index];
  }
}

template <typename Dtype>
void StopLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
 
  StopForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void StopBackward(const int n, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = 0;
  }
}

template <typename Dtype>
void StopLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();

    StopBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(StopLayer);


}  // namespace caffe
